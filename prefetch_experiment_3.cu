#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define CUDACHECK(cmd)                                              \
  do {                                                              \
    hipError_t e = cmd;                                            \
    if (e != hipSuccess) {                                         \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(e));                                \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

constexpr int WARP_SIZE = 32;

__device__ __always_inline half ldg_f16_prefetch_64B(const half *const ptr) {
    half ret;
    asm ("ld.global.L2::64B.b16 %0, [%1];"  : "=h"(*(reinterpret_cast<unsigned short *>(&(ret)))) : "l"(ptr));
    return ret;
}

__device__ __always_inline half ldg_f16_prefetch_128B(const half *const ptr) {
    half ret;
    asm ("ld.global.L2::128B.b16 %0, [%1];"  : "=h"(*(reinterpret_cast<unsigned short *>(&(ret)))) : "l"(ptr));
    return ret;
}

__device__ __always_inline half ldg_f16_prefetch_256B(const half *const ptr) {
    half ret;
    asm ("ld.global.L2::256B.b16 %0, [%1];"  : "=h"(*(reinterpret_cast<unsigned short *>(&(ret)))) : "l"(ptr));
    return ret;
}

__device__ __always_inline float ldg_f32_prefetch_64B(const float *ptr) {
  float ret;
  asm volatile ("ld.global.L2::64B.f32 %0, [%1];"  : "=f"(ret) : "l"(ptr));
  return ret;
}

__device__ __always_inline float ldg_f32_prefetch_128B(const float *ptr) {
  float ret;
  asm volatile ("ld.global.L2::128B.f32 %0, [%1];"  : "=f"(ret) : "l"(ptr));
  return ret;
}

__device__ __always_inline float ldg_f32_prefetch_256B(const float *ptr) {
  float ret;
  asm volatile ("ld.global.L2::256B.f32 %0, [%1];"  : "=f"(ret) : "l"(ptr));
  return ret;
}

__device__ __always_inline float2 ldg_f32v2_prefetch_64B(const float2 *ptr) {
  float2 ret;
  asm volatile ("ld.global.L2::64B.v2.f32 {%0,%1}, [%2];"  : "=f"(ret.x), "=f"(ret.y) : "l"(ptr));
  return ret;
}

__device__ __always_inline float2 ldg_f32v2_prefetch_128B(const float2 *ptr) {
  float2 ret;
  asm volatile ("ld.global.L2::128B.v2.f32 {%0,%1}, [%2];"  : "=f"(ret.x), "=f"(ret.y) : "l"(ptr));
  return ret;
}

__device__ __always_inline float2 ldg_f32v2_prefetch_256B(const float2 *ptr) {
  float2 ret;
  asm volatile ("ld.global.L2::256B.v2.f32 {%0,%1}, [%2];"  : "=f"(ret.x), "=f"(ret.y) : "l"(ptr));
  return ret;
}

__global__
void prefetch_kernel_load_64_prefetch_0(const float* in, float* out) {
  const float2* value_0_ptr = reinterpret_cast<const float2*>(in);
  float2 value_0 = *(value_0_ptr + threadIdx.x);
  __nanosleep(1000000);  // Sleep 1us
  float2 value_1 = value_0_ptr[threadIdx.x + blockDim.x * 1];
  float2 value_2 = value_0_ptr[threadIdx.x + blockDim.x * 2];
  float2 result = make_float2(value_0.x + value_1.x + value_2.x,
                              value_0.y + value_1.y + value_2.y);
  float2* out_ptr = reinterpret_cast<float2*>(out);
  *(out_ptr + threadIdx.x) = result;
}

__global__
void prefetch_kernel_load_64_prefetch_64(const float* in, float* out) {
  const float2* value_0_ptr = reinterpret_cast<const float2*>(in);
  float2 value_0 = ldg_f32v2_prefetch_64B(value_0_ptr + threadIdx.x);
  __nanosleep(1000000);  // Sleep 1us
  float2 value_1 = value_0_ptr[threadIdx.x + blockDim.x * 1];
  float2 value_2 = value_0_ptr[threadIdx.x + blockDim.x * 2];
  float2 result = make_float2(value_0.x + value_1.x + value_2.x,
                              value_0.y + value_1.y + value_2.y);
  float2* out_ptr = reinterpret_cast<float2*>(out);
  *(out_ptr + threadIdx.x) = result;
}

__global__
void prefetch_kernel_load_64_prefetch_128(const float* in, float* out) {
  const float2* value_0_ptr = reinterpret_cast<const float2*>(in);
  float2 value_0 = ldg_f32v2_prefetch_128B(value_0_ptr + threadIdx.x);
  __nanosleep(1000000);  // Sleep 1us
  float2 value_1 = value_0_ptr[threadIdx.x + blockDim.x * 1];
  float2 value_2 = value_0_ptr[threadIdx.x + blockDim.x * 2];
  float2 result = make_float2(value_0.x + value_1.x + value_2.x,
                              value_0.y + value_1.y + value_2.y);
  float2* out_ptr = reinterpret_cast<float2*>(out);
  *(out_ptr + threadIdx.x) = result;
}

__global__
void prefetch_kernel_load_64_prefetch_256(const float* in, float* out) {
  const float2* value_0_ptr = reinterpret_cast<const float2*>(in);
  float2 value_0 = ldg_f32v2_prefetch_256B(value_0_ptr + threadIdx.x);
  __nanosleep(1000000);  // Sleep 1us
  float2 value_1 = value_0_ptr[threadIdx.x + blockDim.x * 1];
  float2 value_2 = value_0_ptr[threadIdx.x + blockDim.x * 2];
  float2 result = make_float2(value_0.x + value_1.x + value_2.x,
                              value_0.y + value_1.y + value_2.y);
  float2* out_ptr = reinterpret_cast<float2*>(out);
  *(out_ptr + threadIdx.x) = result;
}

int launchKernelExperiment() {
  constexpr int page_size = 4096;
  constexpr int nr_element_f32 = 4096 / sizeof(float);
  constexpr int offset = nr_element_f32 / 2;

  void* input;
  CUDACHECK(hipMalloc(&input, page_size));
  CUDACHECK(hipMemset(input, 0, 4096));

  dim3 grid(1, 1, 1);
  dim3 block(16, 1, 1); // Just 16 threads

  const float* in = reinterpret_cast<const float*>(input);
  float* out = reinterpret_cast<float* >(input) + offset;
  std::cout << "In: " << in << "  Out: " << out << std::endl;

  prefetch_kernel_load_64_prefetch_0<<<grid, block>>>(in, out);
  CUDACHECK(hipDeviceSynchronize());
  prefetch_kernel_load_64_prefetch_64<<<grid, block>>>(in, out);
  CUDACHECK(hipDeviceSynchronize());
  prefetch_kernel_load_64_prefetch_128<<<grid, block>>>(in, out);
  CUDACHECK(hipDeviceSynchronize());
  prefetch_kernel_load_64_prefetch_256<<<grid, block>>>(in, out);
  CUDACHECK(hipDeviceSynchronize());

  CUDACHECK(hipFree(input));
  return 0;
}

int main(void) {
  launchKernelExperiment();
  return 0;
}
